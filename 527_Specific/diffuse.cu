#include "hip/hip_runtime.h"
/*
     nvcc -arch compute_70 -code sm_70 diffuse.cu -o diffuse
     ./diffuse /map_8by8_obst12_agents1_ex0.yaml
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include "LoadMap.h"
using namespace std;

// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), (char *)__FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true){
  if (code != hipSuccess)
  {
    fprintf(stderr, "CUDA_SAFE_CALL: %s %s %d\n",
                                       hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

#define IMUL(a, b) __mul24(a, b)

/* structures */
struct point{
    int x;
    int y;
    int length;
    struct point* next;
};

__global__ void gpu_diffuse (int size, data_t* diffuseMap, data_t* obstacleMap, int dx, int dy);
__global__ void gpu_checkDiffusion (int size, data_t* diffuseMap, data_t* obstacleMap, int* checkMap);
int gpu_checkDiffusionHost(int size, int* map);
void cpu_diffuse (int size, data_t* diffuseMap, data_t* obstacleMap, int dx, int dy);
void cpu_rb_diffuse (int size, data_t* diffuseMap, data_t* obstacleMap, int dx, int dy);
int cpu_checkDiffusion(int size, data_t* diffuseMap, data_t* obstacleMap);
void printPath(int num_agents, struct point** paths, int args, char* filein);
int PrintIntArray(int size, int* arr);
int PrintArray(int size, data_t* arr);
double interval(struct timespec start, struct timespec end);
struct point** traversePath(grid* g);


int main(int argc, char *argv[]){
    printf("Start\n");
    char * name;

    printf("\nrowlength, GPU, Kernel, CPU, CPU_RB\n");

	int kal;
	for (kal = 64; kal <= 2048; kal += 64){ // for (kal = 8; kal <= 2048; kal *= 2){
	    int flag = 0;

	    /* retrieve input file */
	    grid* cpuGrid;
	    grid* gpuGrid;
	    grid* cpuBlkGrid;
	    cpuGrid = MakeGrid(kal); //atoi(argv[1])
	    gpuGrid = MakeGrid(kal); //this creates the grid to use
	    cpuBlkGrid = MakeGrid(kal);

	    // Essential variables
	    int size = cpuGrid->size;
	    int mod_size = size + 2;
	    int dx = cpuGrid->dx;
	    int dy = cpuGrid->dy;
	    int it = 0;
	    int fullsize = size * size;

	    // Timing variables
	    struct timespec time_start, time_stop, time2_start, time2_stop;
	    hipEvent_t start, stop;
	    hipEvent_t kernal_start, kernal_stop;
	    float elapsed_cpu, elapsed_cpu_rb, elapsed_gpu, elapsed_kernal;
	    hipEventCreate(&start);
	    hipEventCreate(&stop);
	    hipEventCreate(&kernal_start);
	    hipEventCreate(&kernal_stop);

	    // Select GPU
	    CUDA_SAFE_CALL(hipSetDevice(0));

	    size_t allocSize = mod_size * mod_size * sizeof(data_t);
	    size_t intAllocSize = mod_size * mod_size * sizeof(int);

	    // Allocate host memory
	    data_t *h_dMap;
	    data_t *h_oMap;
	    int *h_dCheck;
	    data_t* diffuseMap = cpuGrid->diff_matrix;
	    data_t* obstacleMap = cpuGrid->obs_matrix;
	    CUDA_SAFE_CALL(hipHostMalloc((void **)&h_dMap, allocSize));
	    CUDA_SAFE_CALL(hipHostMalloc((void **)&h_oMap, allocSize));
	    CUDA_SAFE_CALL(hipHostMalloc((void **)&h_dCheck, intAllocSize));
	    memcpy(h_dMap, diffuseMap, allocSize);
	    memcpy(h_oMap, obstacleMap, allocSize);
	    memset(h_dCheck, 0, mod_size*mod_size*sizeof(int));

	    // Allocate GPU memory
	    data_t *d_dMap;
	    data_t *d_oMap;
	    int *d_dCheck;
	    CUDA_SAFE_CALL(hipMalloc((void **)&d_dMap, allocSize));
	    CUDA_SAFE_CALL(hipMalloc((void **)&d_oMap, allocSize));
	    CUDA_SAFE_CALL(hipMalloc((void **)&d_dCheck, intAllocSize));

	    // Recording timing including data transfers to and from device
	    //printf("Running GPU diffusion...\n");
	    hipEventRecord(start, 0);

	    // Transfer the arrays to the GPU memory
	    CUDA_SAFE_CALL(hipMemcpy(d_dMap, h_dMap, allocSize, hipMemcpyHostToDevice));
	    CUDA_SAFE_CALL(hipMemcpy(d_oMap, h_oMap, allocSize, hipMemcpyHostToDevice));
	    CUDA_SAFE_CALL(hipMemcpy(d_dCheck, h_dCheck, intAllocSize, hipMemcpyHostToDevice));
	    // Defining single block dimensions
	    dim3 dimBlock(16,16);
	    dim3 dimGrid(size/dimBlock.x, size/dimBlock.y);
	    // Run diffusion on GPU
	    hipEventRecord(kernal_start, 0);
	    while(!flag && it < fullsize){
	        it++;
	        for(int i = 0; i < size; i++){
	            //printf("Diffusing...\n");
	            gpu_diffuse<<<dimGrid, dimBlock>>>(mod_size, d_dMap, d_oMap, dx, dy);
	        }

	        //printf("Analyzing diffusion...\n");
	        gpu_checkDiffusion<<<dimGrid, dimBlock>>>(mod_size, d_dMap, d_oMap, d_dCheck);
	        //printf("Transferring diffusion check to host...\n");
	        CUDA_SAFE_CALL(hipMemcpy(h_dCheck, d_dCheck, intAllocSize, hipMemcpyDeviceToHost));
	        //printf("Checking diffusion...\n");
	        flag = gpu_checkDiffusionHost(mod_size, h_dCheck);
	        //printf("Flag: %d\n",flag);
	    }
        hipEventRecord(kernal_stop, 0);
        hipEventSynchronize(stop);
	    // Transfer the results back to the host
	    CUDA_SAFE_CALL(hipMemcpy(h_dMap, d_dMap, allocSize, hipMemcpyDeviceToHost));
	    hipEventRecord(stop,0);
        hipEventSynchronize(stop);
	    hipEventElapsedTime(&elapsed_gpu, start, stop);
	    hipEventElapsedTime(&elapsed_kernal, kernal_start, kernal_stop);
	    hipEventDestroy(start);
	    hipEventDestroy(stop);
	    hipEventDestroy(kernal_start);
	    hipEventDestroy(kernal_stop);
	    //printf("GPU diffusion finished.\n");

	    gpuGrid->diff_matrix = h_dMap;

	    // Compute the results on the host //EDIT
	    //printf("Running CPU diffusion...\n");
	    clock_gettime(CLOCK_REALTIME, &time_start);
	    cpu_diffuse(mod_size, cpuGrid->diff_matrix, cpuGrid->obs_matrix, dx, dy);
	    clock_gettime(CLOCK_REALTIME, &time_stop);
	    //printf("CPU diffusion finished.\n");
	    elapsed_cpu = interval(time_start, time_stop);

	    // Compute the results on the host //EDIT
	    //printf("Running CPU RB diffusion...\n");
	    clock_gettime(CLOCK_REALTIME, &time2_start);
	    cpu_rb_diffuse(mod_size, cpuBlkGrid->diff_matrix, cpuBlkGrid->obs_matrix, dx, dy);
	    clock_gettime(CLOCK_REALTIME, &time2_stop);
	    //printf("CPU RB diffusion finished.\n");
	    elapsed_cpu_rb = interval(time2_start, time2_stop);

	    // Display timing results
	    printf("%d, ", size);
	    printf("%f, ", elapsed_gpu/1000);
	    printf("%f, ", elapsed_kernal/1000);
	    printf("%f, ", elapsed_cpu);
	    printf("%f, \n", elapsed_cpu_rb);

	    // Compare the results //EDIT
	    /*struct point** cpuPaths = traversePath(cpuGrid);
	    struct point** cpuBlkPaths = traversePath(cpuBlkGrid);
	    struct point** gpuPaths = traversePath(gpuGrid);
	    name = "gpu_paths.yaml";
	    printPath(gpuGrid->num_agents, gpuPaths, argc, name);
	    name = "cpu_paths.yaml";
	    printPath(cpuGrid->num_agents, cpuPaths, argc, name);
	    name = "cpu_rb_paths.yaml";
	    printPath(cpuBlkGrid->num_agents, cpuBlkPaths, argc, name);*/

	    // Free-up device and host memory
	    CUDA_SAFE_CALL(hipFree(d_dMap));
	    CUDA_SAFE_CALL(hipFree(d_oMap));
	    CUDA_SAFE_CALL(hipFree(d_dCheck));

	    CUDA_SAFE_CALL(hipHostFree(h_dMap));
	    CUDA_SAFE_CALL(hipHostFree(h_oMap));
	    CUDA_SAFE_CALL(hipHostFree(h_dCheck));

	    free (cpuGrid->diff_matrix);
	    free (cpuGrid->obs_matrix);
	    free (cpuBlkGrid->diff_matrix);
	    free (cpuBlkGrid->obs_matrix);
	}

    return 0;
}

__global__ void gpu_diffuse (int size, data_t* diffuseMap, data_t* obstacleMap, int dx, int dy) {
    int i = blockIdx.y*blockDim.y+threadIdx.y+1;
    int j = blockIdx.x*blockDim.x+threadIdx.x+1;

    data_t newD;
    data_t large = (data_t) size*size*100;
    diffuseMap[dy * size + dx] = large;
    if(i > 0 && j > 0 && i < size-1 && j < size-1){
        newD = (data_t) .25 * (diffuseMap[(i-1)*size + j] + diffuseMap[(i+1)*size + j] + diffuseMap[i*size + j+1] + diffuseMap[i*size + j-1]) * obstacleMap[i*size + j];
        diffuseMap[i*size + j] = newD;
        diffuseMap[dy * size + dx] = large;
    }

}

__global__ void gpu_checkDiffusion (int size, data_t* diffuseMap, data_t* obstacleMap, int* checkMap) {
    int i = blockIdx.y*blockDim.y+threadIdx.y+1;
    int j = blockIdx.x*blockDim.x+threadIdx.x+1;

    checkMap[i*size+j] = !(((int)obstacleMap[i*size+j] && (diffuseMap[i*size+j] > 0)) || (!(int)obstacleMap[i*size+j] && !(diffuseMap[i*size+j] > 0)));

}

int gpu_checkDiffusionHost(int size, int* map){
    long int i, j;
    for (i = 1; i < size-1; i++) {
        for (j = 1; j < size-1; j++) {
            if(map[i*size+j] == 1){return 0;}
        }
    }
    return 1;
}

void cpu_diffuse (int size, data_t* diffuseMap, data_t* obstacleMap, int dx, int dy) {
    long int i, j, it = 0, fullsize = size*size;
    data_t newD;
    data_t large = (data_t) size*size*100;

    while (!cpu_checkDiffusion(size, diffuseMap, obstacleMap) && it < fullsize){
      	it++;
     	for (i = 1; i < size-1; i++) {
          	for (j = 1; j < size-1; j++) {
              	diffuseMap[dy * size + dx] = large;
              	newD = (data_t) .25 * (diffuseMap[(i-1)*size + j] + diffuseMap[(i+1)*size + j] + diffuseMap[i*size + j+1] + diffuseMap[i*size + j-1]) * obstacleMap[i*size + j];
              	diffuseMap[i*size + j] = newD;
            }
        }
    }
}

void cpu_rb_diffuse (int size, data_t* diffuseMap, data_t* obstacleMap, int dx, int dy) {
    long int i, j, alt, over_alt = 1, it = 0, fullsize = size*size;
    data_t newD;
    data_t large = (data_t) size*size*100;

    while (!cpu_checkDiffusion(size, diffuseMap, obstacleMap) && it < fullsize){
        alt = over_alt;
        it++;
        for (i = 1; i < size-1; i++) {
            for (j = alt; j < size-1; j+=2) {
                diffuseMap[dy * size + dx] = large;
                newD = (data_t) .25 * (diffuseMap[(i-1)*size + j] + diffuseMap[(i+1)*size + j] + diffuseMap[i*size + j+1] + diffuseMap[i*size + j-1]) * obstacleMap[i*size + j];
                diffuseMap[i*size + j] = newD;
            }
            alt = (alt == 1) ? 2:1;
        }
        over_alt = (over_alt == 1) ? 2:1;
    }
}

int cpu_checkDiffusion(int size, data_t* diffuseMap, data_t* obstacleMap){
    int i,j;
    for(i = 1; i < size-1; i++){
        for(j = 1; j < size-1; j++){
            if(obstacleMap[i*size+j] && (diffuseMap[i*size+j] == 0)){return 0;}
        }
    }
    return 1;
}

int PrintIntArray(int size, int* arr){
	int i, j, mult;

	for (i = 0; i < size; i++){
		mult = i*size;
		for (j = 0; j < size; j++){
			printf("%d,\t", arr[mult + j]);
		}
		printf("\n");
	}
	printf("\n");
	return 0;
}

int PrintArray(int size, data_t* arr){
	int i, j, mult;

	for (i = 0; i < size; i++){
		mult = i*size;
		for (j = 0; j < size; j++){
			printf("%.4f,\t", arr[mult + j]);
		}
		printf("\n");
	}
	printf("\n");
	return 0;
}

struct point** traversePath(grid* g){

    int rowlen = g->size;
    int row_bound = rowlen + 1;
    int mod_size = rowlen + 2;
    int num_agents = g->num_agents;
    data_t *data = g->diff_matrix;
    struct agent* a = g->agents;
    struct point* pt_it;
    int i;
    int x, y, arrsize = rowlen*rowlen;
    data_t currentSpot, target;
    data_t neighborMax;
    int neighborIter;
    int pathLength;
    target = g->dy*mod_size + g->dx;

    struct point** paths = (struct point**) calloc(num_agents, sizeof(struct point*));

    for (i = 0; i < num_agents; i++){

        x = a->sx;
        y = a->sy;
        pathLength = 0;
        paths[i] = (struct point*) calloc(1, sizeof(struct point));
        paths[i]->x = x;
        paths[i]->y = y;
        paths[i]->next = NULL;
        pt_it = paths[i];
        currentSpot = y*mod_size + x;

        //printf("AGENT%d: [%d, %d]\n", i, x, y);

        while(currentSpot != target){

            pt_it->next = (struct point*) calloc(1, sizeof(struct point));

            neighborMax = 0;
            if(data[(y-1)*mod_size + x] > neighborMax){
                neighborMax = data[(y-1)*mod_size+x];
                neighborIter = 1;
            }
            if(data[(y+1)*mod_size + x] > neighborMax){
                neighborMax = data[(y+1)*mod_size+x];
                neighborIter = 2;
            }
            if(data[y*mod_size+x+1] > neighborMax){
                neighborMax = data[y*mod_size+x+1];
                neighborIter = 3;
            }
            if(data[y*mod_size+x-1] > neighborMax){
                neighborMax = data[y*mod_size+x-1];
                neighborIter = 4;
            }

            if(neighborIter == 1) y--;
            else if(neighborIter == 2) y++;
            else if(neighborIter == 3) x++;
            else if(neighborIter == 4) x--;

            // setup for next iteration
            currentSpot = y*mod_size+x;
            pt_it = pt_it->next;
            pt_it->x = x;
            pt_it->y = y;
            pt_it->next = NULL;

            pathLength++;
        }

        paths[i]->length = pathLength;
        a = a->next;
    }

    return paths;
}

void printPath(int num_agents, struct point** paths, int args, char* filein)
{
    int i;
    struct point* pt_it;
    FILE *fp;
    char* new_name;

    // get new file name
    new_name = filein;

    fp = fopen(new_name,"w+");
    fprintf(fp, "paths:\n");

    for (i = 0; i < num_agents; i++){
        fprintf(fp, "-   name: agent%d\n", i);
        pt_it = paths[i];
        fprintf(fp, "    path:\n");
        while (pt_it != NULL){
            fprintf(fp, "    - [%d,%d]\n", pt_it->x -1, pt_it->y -1);
            pt_it = pt_it->next;
        }
        fprintf(fp, "    length: %d\n", paths[i]->length);
    }

    fclose(fp);
}

double interval(struct timespec start, struct timespec end)
{
  struct timespec temp;
  temp.tv_sec = end.tv_sec - start.tv_sec;
  temp.tv_nsec = end.tv_nsec - start.tv_nsec;
  if (temp.tv_nsec < 0) {
    temp.tv_sec = temp.tv_sec - 1;
    temp.tv_nsec = temp.tv_nsec + 1000000000;
  }
  return (((double)temp.tv_sec) + ((double)temp.tv_nsec)*1.0e-9);
}
